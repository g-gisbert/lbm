#include "hip/hip_runtime.h"
#include "LBMGPU.cuh"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <chrono>



__constant__ float d_weights[9], d_e[18];

LBMGPU::LBMGPU(int w, int h) :
    m_width(w), m_height(h), m_blockSize(dim3{16,16}),
    m_gridSize(dim3{(w + m_blockSize.x - 1) / m_blockSize.x, (h + m_blockSize.y - 1) / m_blockSize.y}),
    h_velU(new float[m_width * m_height]), h_velV(new float[m_width * m_height]), h_ink(new float[m_width * m_height * 3]) {

    hipMalloc((void**)&d_oldDensities, w * h * 9 * sizeof(float));
    hipMalloc((void**)&d_densities, w * h * 9 * sizeof(float));
    hipMalloc((void**)&d_velU, w * h * sizeof(float));
    hipMalloc((void**)&d_velV, w * h * sizeof(float));
    hipMalloc((void**)&d_walls, w * h * sizeof(uint8_t));
    hipMalloc((void**)&d_ink, 3 * w * h * sizeof(float));
    hipMalloc((void**)&d_oldInk, 3 * w * h * sizeof(float));

    hipMemset(d_oldDensities, 0, w * h * 9 * sizeof(float));
    hipMemset(d_densities, 0, w * h * 9 * sizeof(float));
    hipMemset(d_velU, 0, w * h * sizeof(float));
    hipMemset(d_velV, 0, w * h * sizeof(float));
    hipMemset(d_ink, 0, 3 * w * h * sizeof(float));
    hipMemset(d_oldInk, 0, 3 * w * h * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(d_weights), h_weights, 9 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_e), h_e, 18 * sizeof(float));

    initializeWalls<<<m_gridSize, m_blockSize>>>(d_walls, h, w);
    hipDeviceSynchronize();
    initializeDensities<<<m_gridSize, m_blockSize>>>(d_densities, h, w);
    hipDeviceSynchronize();
}

LBMGPU::~LBMGPU() {
    hipFree(d_oldDensities);
    hipFree(d_densities);
    hipFree(d_velU);
    hipFree(d_velV);
    hipFree(d_walls);
    hipFree(d_ink);
    hipFree(d_oldInk);
    delete[] h_velU;
    delete[] h_velV;
    delete[] h_ink;
}


void LBMGPU::step(SimulationParams& params, unsigned char* buffer) {

    for (int iter = 0; iter < 10; ++iter) {

        swapPointers(d_oldDensities, d_densities);
        swapPointers(d_oldInk, d_ink);

        addInk<<<m_gridSize, m_blockSize>>>(d_oldInk, m_height, m_width);

        advection<<<m_gridSize, m_blockSize>>>(d_densities, d_oldDensities, m_height, m_width);
        hipDeviceSynchronize();

        collision<<<m_gridSize, m_blockSize>>>(params.invTau, params.speed, d_densities, d_velU,
                                                               d_velV, d_walls, d_ink, d_oldInk, m_height, m_width);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_velU, d_velU, m_width * m_height * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_velV, d_velV, m_width * m_height * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_ink, d_ink, m_width * m_height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < m_height; ++i) {
        for (int j = 0; j < m_width; ++j) {
            float inkR = h_ink[i * m_width * 3 + j * 3 + 0] * 255;
            float inkG = h_ink[i * m_width * 3 + j * 3 + 1] * 255;
            float inkB = h_ink[i * m_width * 3 + j * 3 + 2] * 255;
            float velU = h_velU[i * m_width + j];
            float velV = h_velV[i * m_width + j];
            float value = sqrtf(velU * velU + velV * velV) * 5.0 * 255;

            if (params.mode) {
                buffer[3 * (i * m_width + j)] = value / 4;
                buffer[3 * (i * m_width + j) + 1] = value / 2;
                buffer[3 * (i * m_width + j) + 2] = value;
            }
            else {
                buffer[3 * (i * m_width + j)] = inkR;
                buffer[3 * (i * m_width + j) + 1] = inkG;
                buffer[3 * (i * m_width + j) + 2] = inkB;
            }
        }
    }
}


__global__ void initializeWalls(unsigned char* array, unsigned int rows, unsigned int cols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        if ((row - rows/2)*(row - rows/2) + (col - 320)*(col - 320) < 105*105)
            array[row * cols + col] = 255;
        else if ((row - 0.75*rows)*(row - 0.75*rows) + (col - 200)*(col - 200) < 25*25)
            array[row * cols + col] = 255;
        else if ((row - 0.25*rows)*(row - 0.25*rows) + (col - 200)*(col - 200) < 25*25)
            array[row * cols + col] = 255;
        else
            array[row * cols + col] = 0;
    }
}

__global__ void initializeDensities(float* d_densities, unsigned int rows, unsigned int cols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {

        float v =  0.1f * (1.0f + 0.1f * float(row) / float(rows));
        float lastTerm = -1.5f * v*v;
        for (int k = 0; k < 9; ++k) {
            float eiu = d_e[2*k+1] * v;
            float feq = d_weights[k] * (1.0f + 3.0f * eiu + 4.5f * eiu * eiu + lastTerm);
            d_densities[row * cols * 9 + col * 9 + k] = feq;
        }
    }
}

void swapPointers(float*& array1, float*& array2) {
    float *tmp = array1;
    array1 = array2;
    array2 = tmp;
}

__global__ void addInk(float* d_ink, unsigned int rows, unsigned int cols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        if (row >= 140+rows/2 && row < 200+rows/2) {
            d_ink[row * cols * 3 + 0 * 3 + 0] = 1.0f;
            d_ink[row * cols * 3 + 0 * 3 + 1] = 0.0f;
            d_ink[row * cols * 3 + 0 * 3 + 2] = 0.2f;
        }
        if (row < rows/2-140 && row >= rows/2-200) {
            d_ink[row * cols * 3 + 0 * 3 + 0] = 0.0f;
            d_ink[row * cols * 3 + 0 * 3 + 1] = 0.2f;
            d_ink[row * cols * 3 + 0 * 3 + 2] = 1.0f;
        }
        if (row >= rows/2-25 && row < 25+rows/2) {
            d_ink[row * cols * 3 + 0 * 3 + 0] = 0.2f;
            d_ink[row * cols * 3 + 0 * 3 + 1] = 1.0f;
            d_ink[row * cols * 3 + 0 * 3 + 2] = 0.2f;
        }
    }
}

__global__ void advection(float* d_densities, const float* d_oldDensities, unsigned int rows, unsigned int cols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        unsigned int xplus = ((col == cols - 1) ? (0) : (col + 1));
        unsigned int xminus = ((col == 0) ? (cols - 1) : (col - 1));
        unsigned int yplus = ((row == rows - 1) ? (0) : (row + 1));
        unsigned int yminus = ((row == 0) ? (rows - 1) : (row - 1));
        d_densities[row * cols * 9 + col * 9 + 0] = d_oldDensities[row * cols * 9 + col * 9 + 0];
        d_densities[row * cols * 9 + col * 9 + 1] = d_oldDensities[row * cols * 9 + xminus * 9 + 1];
        d_densities[row * cols * 9 + col * 9 + 2] = d_oldDensities[yminus * cols * 9 + col * 9 + 2];
        d_densities[row * cols * 9 + col * 9 + 3] = d_oldDensities[row * cols * 9 + xplus * 9 + 3];
        d_densities[row * cols * 9 + col * 9 + 4] = d_oldDensities[yplus * cols * 9 + col * 9 + 4];
        d_densities[row * cols * 9 + col * 9 + 5] = d_oldDensities[yminus * cols * 9 + xminus * 9 + 5];
        d_densities[row * cols * 9 + col * 9 + 6] = d_oldDensities[yminus * cols * 9 + xplus * 9 + 6];
        d_densities[row * cols * 9 + col * 9 + 7] = d_oldDensities[yplus * cols * 9 + xplus * 9 + 7];
        d_densities[row * cols * 9 + col * 9 + 8] = d_oldDensities[yplus * cols * 9 + xminus * 9 + 8];
    }
}

__global__ void collision(float invTau, float speed, float* d_densities, float* d_velU, float* d_velV, unsigned char* d_walls,
                          float* d_ink, float* d_oldInk, unsigned int rows, unsigned int cols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        // Rho
        float rho = d_densities[row * cols * 9 + col * 9 + 0] + d_densities[row * cols * 9 + col * 9 + 1] +
                    d_densities[row * cols * 9 + col * 9 + 2] + d_densities[row * cols * 9 + col * 9 + 3] +
                    d_densities[row * cols * 9 + col * 9 + 4] + d_densities[row * cols * 9 + col * 9 + 5] +
                    d_densities[row * cols * 9 + col * 9 + 6] + d_densities[row * cols * 9 + col * 9 + 7] +
                    d_densities[row * cols * 9 + col * 9 + 8];

        if (d_walls[row * cols + col] > 127) {
            rho = 1.0f;
        }

        // Update velocities
        float velU = (d_densities[row * cols * 9 + col * 9 + 2] - d_densities[row * cols * 9 + col * 9 + 4] +
                      d_densities[row * cols * 9 + col * 9 + 5] + d_densities[row * cols * 9 + col * 9 + 6] -
                      d_densities[row * cols * 9 + col * 9 + 7] - d_densities[row * cols * 9 + col * 9 + 8]) / rho;
        float velV = (d_densities[row * cols * 9 + col * 9 + 1] - d_densities[row * cols * 9 + col * 9 + 3] +
                      d_densities[row * cols * 9 + col * 9 + 5] - d_densities[row * cols * 9 + col * 9 + 6] -
                      d_densities[row * cols * 9 + col * 9 + 7] + d_densities[row * cols * 9 + col * 9 + 8]) / rho;

        float norm = sqrtf(velU*velU + velV*velV);
        if (norm > 0.2f) {
            velU *= 0.2f / norm;
            velV *= 0.2f / norm;
        }
        if (col == 0 || col == cols-1) {
            rho = 1.0f;
            velU = 0.0f;
            velV = 0.1f;
        }
        if (d_walls[row * cols + col] > 127) {
            velU = 0.0f;
            velV = 0.0f;
        }
        d_velU[row * cols + col] = velU;
        d_velV[row * cols + col] = velV;

        // Ink
        float destRow = float(col) - 1.0f * velV * speed;
        float destColumn = float(row) - 1.0f * velU * speed;
        float wj = destRow - floorf(destRow);
        float wi = destColumn - floorf(destColumn);
        int jDest = kerPositiveFmod(floorf(destRow), (cols - 1));
        int iDest = kerPositiveFmod(floorf(destColumn), (rows - 1));

        float r = (1.0f - wi)*(1.0f - wj)*d_oldInk[iDest * cols * 3 + jDest * 3 + 0] +
                  (1.0f - wi)*wj*d_oldInk[iDest * cols * 3 + (jDest+1) * 3 + 0] +
                  wi*(1.0f - wj)*d_oldInk[(iDest+1) * cols * 3 + jDest * 3 + 0] +
                  wi*wj*d_oldInk[(iDest+1) * cols * 3 + (jDest+1) * 3 + 0];
        float g = (1.0f - wi)*(1.0f - wj)*d_oldInk[iDest * cols * 3 + jDest * 3 + 1] +
                  (1.0f - wi)*wj*d_oldInk[iDest * cols * 3 + (jDest+1) * 3 + 1] +
                  wi*(1.0f - wj)*d_oldInk[(iDest+1) * cols * 3 + jDest * 3 + 1] +
                  wi*wj*d_oldInk[(iDest+1) * cols * 3 + (jDest+1) * 3 + 1];
        float b = (1.0f - wi)*(1.0f - wj)*d_oldInk[iDest * cols * 3 + jDest * 3 + 2] +
                  (1.0f - wi)*wj*d_oldInk[iDest * cols * 3 + (jDest+1) * 3 + 2] +
                  wi*(1.0f - wj)*d_oldInk[(iDest+1) * cols * 3 + jDest * 3 + 2] +
                  wi*wj*d_oldInk[(iDest+1) * cols * 3 + (jDest+1) * 3 + 2];
        if (r > 1.0)
            r = 1.0;
        if (g > 1.0)
            g = 1.0;
        if (b > 1.0)
            b = 1.0;
        d_ink[row * cols * 3 + col * 3 + 0] = r*0.9999f;
        d_ink[row * cols * 3 + col * 3 + 1] = g*0.9999f;
        d_ink[row * cols * 3 + col * 3 + 2] = b*0.9999f;

        // Collision eq
        float lastTerm = - 1.5f*(velU*velU + velV*velV);
        for (int k = 0; k < 9; ++k) {
            float eiu = d_e[2*k]*velU + d_e[2*k+1]*velV;
            float feq = rho * d_weights[k] * (1.0f + 3.0f*eiu + 4.5f*eiu*eiu + lastTerm);
            d_densities[row * cols * 9 + col * 9 + k] += invTau * (feq - d_densities[row * cols * 9 + col * 9 + k]);

            if (d_walls[row * cols + col] > 127 || col == 0 || col == cols-1) {
                d_densities[row * cols * 9 + col * 9 + k] = feq;
            }
        }
    }
}

__device__ float kerPositiveFmod(float val, float mod) {
    float ans = fmodf(val, mod);
    if (ans < 0.0f)
        ans += mod;
    return ans;
}
